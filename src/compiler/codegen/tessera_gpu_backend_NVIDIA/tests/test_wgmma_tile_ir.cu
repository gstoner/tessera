#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cstdio>
#include <vector>

// Forward declare from ptx_wgmma_bf16.cu
extern "C" void launch_wgmma_bf16_gemm(const __hip_bfloat16*, const __hip_bfloat16*, float*,
                                       int,int,int, float,float, hipStream_t);

static __hip_bfloat16 b16(float x){ return __float2bfloat16(x); }

int main() {
    int M=128,N=128,K=128;
    std::vector<__hip_bfloat16> A(M*K), B(K*N);
    std::vector<float> C(M*N, 1.0f);
    for (int i=0;i<M*K;i++) A[i] = b16((i%7 - 3)/4.0f);
    for (int i=0;i<K*N;i++) B[i] = b16(((i*5)%11 - 5)/6.0f);
    __hip_bfloat16 *dA,*dB; float* dC;
    hipMalloc(&dA, sizeof(__hip_bfloat16)*M*K);
    hipMalloc(&dB, sizeof(__hip_bfloat16)*K*N);
    hipMalloc(&dC, sizeof(float)*M*N);
    hipMemcpy(dA, A.data(), sizeof(__hip_bfloat16)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB, B.data(), sizeof(__hip_bfloat16)*K*N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C.data(), sizeof(float)*M*N, hipMemcpyHostToDevice);

    launch_wgmma_bf16_gemm(dA,dB,dC,M,N,K,1.0f,0.0f,0);
    hipMemcpy(C.data(), dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    double sum=0; for (auto v: C) sum += v;
    std::printf("WGMMA placeholder checksum: %.6f\n", sum);
    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}
