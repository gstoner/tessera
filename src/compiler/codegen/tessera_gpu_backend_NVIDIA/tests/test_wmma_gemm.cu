#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cstdio>
#include <vector>
#include <random>
#include <cassert>
#include "tessera/gpu/target.h"

static __half h(float x){ return __float2half(x); }
static __hip_bfloat16 b16(float x){ return __float2bfloat16(x); }

int main() {
    using namespace tessera::gpu;
    TesseraGpuBackend be;
    if (!be.cudaAvailable()) { std::printf("No CUDA device.\n"); return 0; }
    std::printf("SM version: %d\n", be.smVersion());

    int M=128,N=128,K=128;
    std::vector<__half> A(M*K), B(K*N);
    std::vector<float>  Cf(M*N, 0.0f);
    for (int i=0;i<M*K;i++) A[i] = h((i%13 - 6) / 7.0f);
    for (int i=0;i<K*N;i++) B[i] = h(((i*3)%17 - 8) / 9.0f);

    __half* dA; __half* dB; float* dC;
    hipMalloc(&dA, sizeof(__half)*M*K);
    hipMalloc(&dB, sizeof(__half)*K*N);
    hipMalloc(&dC, sizeof(float)*M*N);
    hipMemcpy(dA, A.data(), sizeof(__half)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB, B.data(), sizeof(__half)*K*N, hipMemcpyHostToDevice);
    hipMemset(dC, 0, sizeof(float)*M*N);

    be.wmma_gemm_fp16(dA, dB, dC, M,N,K, 1.0f, 0.0f);
    hipMemcpy(Cf.data(), dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    // Just print a checksum
    double sum=0; for (auto v: Cf) sum += v;
    std::printf("FP16 GEMM checksum: %.6f\n", sum);

    // BF16 (if sm80+)
    if (be.smVersion() >= 80) {
        std::vector<__hip_bfloat16> Ab(M*K), Bb(K*N);
        for (int i=0;i<M*K;i++) Ab[i] = b16((i%11 - 5) / 7.0f);
        for (int i=0;i<K*N;i++) Bb[i] = b16(((i*5)%19 - 9) / 9.0f);
        __hip_bfloat16 *dAb,*dBb; hipMalloc(&dAb, sizeof(__hip_bfloat16)*M*K);
        hipMalloc(&dBb, sizeof(__hip_bfloat16)*K*N);
        hipMemcpy(dAb, Ab.data(), sizeof(__hip_bfloat16)*M*K, hipMemcpyHostToDevice);
        hipMemcpy(dBb, Bb.data(), sizeof(__hip_bfloat16)*K*N, hipMemcpyHostToDevice);
        hipMemset(dC, 0, sizeof(float)*M*N);
        be.wmma_gemm_bf16(dAb, dBb, dC, M,N,K, 1.0f, 0.0f);
        hipMemcpy(Cf.data(), dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);
        double sum2=0; for (auto v: Cf) sum2 += v;
        std::printf("BF16 GEMM checksum: %.6f\n", sum2);
        hipFree(dAb); hipFree(dBb);
    }

    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}
